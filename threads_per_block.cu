#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    return 0;
}

