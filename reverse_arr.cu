#include <hip/hip_runtime.h>
#include <iostream>

__global__ void reverseArray(float *arr, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = N - i - 1; // Mirror index

    if (i < j) { // Only process the first half
        float temp = arr[i];
        arr[i] = arr[j];
        arr[j] = temp;
    }
}

void reverseArrayHost(float *arr, int N) {
    float *d_arr;
    size_t size = N * sizeof(float);

    hipMalloc(&d_arr, size);
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    reverseArray<<<blocksPerGrid, threadsPerBlock>>>(d_arr, N);

    hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    const int N = 10;
    float arr[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};

    std::cout << "Original Array: ";
    for (int i = 0; i < N; i++) std::cout << arr[i] << " ";
    std::cout << std::endl;

    reverseArrayHost(arr, N);

    std::cout << "Reversed Array: ";
    for (int i = 0; i < N; i++) std::cout << arr[i] << " ";
    std::cout << std::endl;

    return 0;
}
