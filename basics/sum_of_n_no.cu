
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
    printf("%d ", C[i]);    
}

int main() {
    int N = 1 << 20; // 1 Million elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize vectors and calculate their sums
    float sum_A = 0, sum_B = 0;
    for(int i = 0; i < N; i++) {
        h_A[i] = float(i);
        h_B[i] = float(i * 2);
        sum_A += h_A[i];
        sum_B += h_B[i];
    }

    // Print the sums of the elements of A and B
    printf("Sum of vector A: %f\n", sum_A);
    printf("Sum of vector B: %f\n", sum_B);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result
    for(int i = 0; i < N; i++) {
        if(h_C[i] != h_A[i] + h_B[i]) {
            printf("Error at index %d: %f != %f + %f\n", i, h_C[i], h_A[i], h_B[i]);
            break;
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Vector addition completed successfully.\n");
    return 0;
}
