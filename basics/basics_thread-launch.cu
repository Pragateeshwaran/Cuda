#include<iostream>
#include<hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    kernel<<<1, 1024>>>(); // if 1025 will not produce any output because the kernel have only 1024 threads
    hipDeviceSynchronize();
    kernel<<<1, 1025>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }

    return 0;
}   