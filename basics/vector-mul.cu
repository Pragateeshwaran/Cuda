#include <hip/hip_runtime.h>
#include <stdio.h>
int main() {
    float *d_array;
    hipError_t err = hipMalloc(&d_array, 1024 * sizeof(float));
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("Last error: %s\n", hipGetErrorString(hipGetLastError()));
    hipFree(d_array);
    return 0;
}